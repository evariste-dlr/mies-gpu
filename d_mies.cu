#include "hip/hip_runtime.h"
#include <iostream>
#include "d_mies.h"

#define TH_NODES_PER_BLOCK 128
#define TH_EDGES_PER_BLOCK 128
#define EDGES_PER_THREAD 1
#define NODES_PER_THREAD 1

float GPU_ELAPSED_TIME=-1.0;


void cudaTimerStart(hipEvent_t &start) {
        hipEventCreate(&start);
        hipEventRecord(start, 0 );
}

float cudaTimerStop(hipEvent_t &start) {
        hipEvent_t stop;
        hipEventCreate(&stop);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float time;
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return time;
}


__global__
void d_mies_min_row( const sparse_mat* adj,
                     float* mins,
                     uint rows )
{
  uint n = blockDim.x * blockIdx.x + threadIdx.x;

  if (n < rows){

    float min = INFTY;
    for (int i=adj->csr_row_ptr[n]; i<adj->csr_row_ptr[n+1]; i++)
      if (min > adj->val[i])
        min = adj->val[i];

    mins[n] = min;
  }
}


__global__
void d_mies_survivor_round( const sparse_mat * adj,
                            sparse_mat * surv,
                            const float* mins,
                            uint nnz,
                            const uint * n_candidates )
{
  uint e = blockDim.x * blockIdx.x + threadIdx.x;

  // If e is candidate
  if (e < nnz){
   if(adj->val[e] != INFTY){

    int i = adj->coo_row_ind[e];
    int j = adj->coo_col_ind[e];

    float local_min = (mins[i] < mins[j]) ? mins[i] : mins[j];

    // Edge e is a survivor if: (e is a survivor) OR (e is a local min)
    if (local_min == adj->val[e])
      surv->val[e] = 1.0;
   }
  }
}


__global__
void d_mies_surv_row( const sparse_mat* surv,
                      bool* exist_surv,
                      uint rows )
{
  uint n = blockDim.x * blockIdx.x + threadIdx.x;

  if (n < rows){

    bool exist = exist_surv[n];
    for ( int i=surv->csr_row_ptr[n];
          i<surv->csr_row_ptr[n+1] && !exist;
          i++ )
      exist = surv->val[i] > 0.0;

    exist_surv[n] = exist;
  }
}


__global__
void d_mies_candidate_round( sparse_mat * adj,
                             const sparse_mat * surv,
                             const bool* exist_surv,
                             uint nnz,
                             uint * n_candidates )
{
  uint e = blockDim.x * blockIdx.x + threadIdx.x;

  // If e is candidate
  if (e < nnz){
   if (adj->val[e] != INFTY){

    int i = adj->coo_row_ind[e];
    int j = adj->coo_col_ind[e];

    if (exist_surv[i] || exist_surv[j]){
      adj->val[e] = INFTY;
      atomicSub(n_candidates, 1);
    }
   }
  }
}



void d_mies( sparse_mat & adj, sparse_mat & surv )
{
  hipSetDevice(0);

  hipEvent_t start;
  //cudaTimerStart(start);


  /* ======== Upload the matrices onto the GPU ======== */
  
  uint n_candidates = adj.nnz;
  uint* d_n_candidates;
  hipMalloc( &d_n_candidates, sizeof(uint) );
  hipMemcpy( d_n_candidates, &n_candidates, sizeof(uint), hipMemcpyHostToDevice );

  float* d_adj_val;
  uint* d_adj_coo_row_ind;
  uint* d_adj_coo_col_ind;
  uint* d_adj_csr_row_ptr;
  float* d_surv_val;
  uint* d_surv_coo_row_ind;
  uint* d_surv_coo_col_ind;
  uint* d_surv_csr_row_ptr;

  hipMalloc(&d_adj_val, adj.nnz * sizeof(float));
  hipMalloc(&d_adj_coo_row_ind, adj.nnz * sizeof(uint));
  hipMalloc(&d_adj_coo_col_ind, adj.nnz * sizeof(uint));
  hipMalloc(&d_adj_csr_row_ptr, (adj.rows+1) * sizeof(uint));

  hipMalloc(&d_surv_val, surv.nnz * sizeof(float));
  hipMalloc(&d_surv_coo_row_ind, surv.nnz * sizeof(uint));
  hipMalloc(&d_surv_coo_col_ind, surv.nnz * sizeof(uint));
  hipMalloc(&d_surv_csr_row_ptr, (surv.rows+1) * sizeof(uint));

  float* d_mins;
  bool* d_exist_surv;
  hipMalloc(&d_mins, adj.rows * sizeof(float));
  hipMemset( d_mins, 0.0, adj.rows * sizeof(float));
  hipMalloc(&d_exist_surv, adj.rows * sizeof(bool));
  hipMemset( d_exist_surv, 0, adj.rows * sizeof(bool));

  sparse_mat * d_adj;
  sparse_mat * d_surv;
  hipMalloc(&d_adj, sizeof(sparse_mat));
  hipMalloc(&d_surv, sizeof(sparse_mat));

  // Bind the addresses
  sparse_mat h_adj_bind = {
    .val = d_adj_val,
    .coo_row_ind = d_adj_coo_row_ind,
    .coo_col_ind = d_adj_coo_col_ind,
    .csr_row_ptr = d_adj_csr_row_ptr,
    .nnz = adj.nnz,
    .rows = adj.rows,
    .cols = adj.cols
  };

  sparse_mat h_surv_bind = {
    .val = d_surv_val,
    .coo_row_ind = d_surv_coo_row_ind,
    .coo_col_ind = d_surv_coo_col_ind,
    .csr_row_ptr = d_surv_csr_row_ptr,
    .nnz = surv.nnz,
    .rows = surv.rows,
    .cols = surv.cols
  };

  hipMemcpy( d_adj, &h_adj_bind, sizeof(sparse_mat), hipMemcpyHostToDevice );
  hipMemcpy( d_surv, &h_surv_bind, sizeof(sparse_mat), hipMemcpyHostToDevice );

  hipMemcpy( d_adj_val, adj.val, adj.nnz*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_adj_coo_row_ind, adj.coo_row_ind, adj.nnz*sizeof(uint), hipMemcpyHostToDevice );
  hipMemcpy( d_adj_coo_col_ind, adj.coo_col_ind, adj.nnz*sizeof(uint), hipMemcpyHostToDevice );
  hipMemcpy( d_adj_csr_row_ptr, adj.csr_row_ptr, (adj.rows+1)*sizeof(uint), hipMemcpyHostToDevice );

  hipMemcpy( d_surv_val, surv.val, surv.nnz*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy( d_surv_coo_row_ind, surv.coo_row_ind, surv.nnz*sizeof(uint), hipMemcpyHostToDevice );
  hipMemcpy( d_surv_coo_col_ind, surv.coo_col_ind, surv.nnz*sizeof(uint), hipMemcpyHostToDevice );
  hipMemcpy( d_surv_csr_row_ptr, surv.csr_row_ptr, (surv.rows+1)*sizeof(uint), hipMemcpyHostToDevice );


  /* ======== Main Loop  ======== */

  dim3 dimBlockNodes(TH_NODES_PER_BLOCK, 1, 1);
  dim3 dimGridNodes(ceil((double)(adj.rows) / (double)TH_NODES_PER_BLOCK*NODES_PER_THREAD), 1, 1);
  dim3 dimBlockEdges(TH_EDGES_PER_BLOCK, 1, 1);
  dim3 dimGridEdges(ceil((double)(adj.nnz) / ((double)TH_EDGES_PER_BLOCK)*EDGES_PER_THREAD), 1, 1);

  cudaTimerStart(start);

  bool* h_exist_surv = (bool*) calloc(adj.rows, sizeof(bool));
  bool* h_eq_surv = (bool*) calloc(adj.rows, sizeof(bool));

  while (n_candidates > 0){
    d_mies_min_row<<<dimGridNodes, dimBlockNodes>>>( d_adj, d_mins, adj.rows );
    d_mies_survivor_round<<<dimGridEdges, dimBlockEdges>>>( d_adj, d_surv, d_mins, adj.nnz, d_n_candidates );

    d_mies_surv_row<<<dimGridNodes, dimBlockNodes>>>( d_surv, d_exist_surv, surv.rows );
    d_mies_candidate_round<<<dimGridEdges, dimBlockEdges>>>( d_adj, d_surv, d_exist_surv, adj.nnz, d_n_candidates );

    hipMemcpy( &n_candidates, d_n_candidates, sizeof(uint), hipMemcpyDeviceToHost );
  }

  hipMemcpy( surv.val, d_surv_val, surv.nnz*sizeof(float), hipMemcpyDeviceToHost );

  GPU_ELAPSED_TIME = cudaTimerStop(start);


  /* ======== Clean up  ======== */

  hipFree(d_adj);
  hipFree(d_surv);

  hipFree(d_mins);
  hipFree(d_exist_surv);

  hipFree(d_adj_val);
  hipFree(d_adj_coo_row_ind);
  hipFree(d_adj_coo_col_ind);
  hipFree(d_adj_csr_row_ptr);

  hipFree(d_surv_val);
  hipFree(d_surv_coo_row_ind);
  hipFree(d_surv_coo_col_ind);
  hipFree(d_surv_csr_row_ptr);

  hipFree(d_n_candidates);
}
